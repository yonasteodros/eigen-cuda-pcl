#include "hip/hip_runtime.h"
#include <cudakernel.cuh>
#include <iostream>
#include <stdio.h>
#include <eigen3/Eigen/Core>

static void HandleError( hipError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != hipSuccess)
    {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// CUDA Version
namespace CudaKernel
{
    __global__ void cu_dot(Eigen::Vector4f *v1, Eigen::Vector4f *c1, pcl::gpu::PtrSz<pcl::PointXYZRGB> output, size_t N)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;


        if(idx < N)
        {
            output[idx].x = v1[idx].x();
            output[idx].y = v1[idx].y();
            output[idx].z = v1[idx].z();


            unsigned char r = int(c1[idx].x()) >> 16 & 0xFF; /* gets 2nd MSB 0x06 */
            unsigned char g = int(c1[idx].y()) >> 8 & 0xFF; /* gets the 2nd LSB 0xd3 */
            unsigned char b = int(c1[idx].z()) & 0xFF; /* gets the LSB 0xb0 */
            uint32_t rgb = ((uint32_t)r << 16 | (uint32_t)g << 8 | (uint32_t)b);

            output[idx].rgb=*reinterpret_cast<float*>(&rgb);

        }

        return;
    }

    // The wrapper for the calling of the actual kernel
    pcl::PointCloud<pcl::PointXYZRGB> dot(const std::vector<Eigen::Vector4f> & v1, const std::vector<Eigen::Vector4f> & c1, const pcl::PointCloud<pcl::PointXYZRGB> & output)
    {
        int n = v1.size();
        double *ret = new double[n];
        pcl::PointCloud<pcl::PointXYZRGB> cloud;
        cloud.width  = n;
        cloud.height = 1;
        cloud.is_dense=false;
        cloud.points.resize (cloud.width * cloud.height);

        // Allocate device arrays
        Eigen::Vector4f *dev_v1;
        Eigen::Vector4f *dev_c1;
        HANDLE_ERROR(hipMalloc((void **)&dev_v1, sizeof(Eigen::Vector4f)*n));
        HANDLE_ERROR(hipMalloc((void **)&dev_c1, sizeof(Eigen::Vector4f)*n));

        pcl::gpu::DeviceArray<pcl::PointXYZRGB> dev_pcl;
        dev_pcl.create(n);
        //pcl :: gpu :: DeviceArray <pcl :: PointXYZ> DevicePointArray (n);//allocation the new GPU memory!

        HANDLE_ERROR(hipMalloc((void **)&dev_pcl, sizeof(pcl::PointXYZRGB)*n));

        // Copy to device
        HANDLE_ERROR(hipMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector4f)*n, hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(dev_c1, c1.data(), sizeof(Eigen::Vector4f)*n, hipMemcpyHostToDevice));

        cu_dot<<<(n+1023)/1024, 1024>>>(dev_v1, dev_c1, dev_pcl, n);

        dev_pcl.download(cloud.points);

        return cloud;
    }
}

